#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"

__global__ void aplusb_matrix_bad(const unsigned int* a,
                       const unsigned int* b,
                             unsigned int* c,
                             unsigned int  width,
                             unsigned int  height)
{
    // все три массива - линейно выложенные двумерные матрицы размера width (число столбиков) x height (число рядов)
    // при этом в памяти подряд идут элементы являющимися соседями в рамках одного ряда,
    // т.е. матрица выложена в памяти линейно ряд за рядом
    // т.е. если в матрице сделать шаг вправо или влево на одну ячейку - то в памяти мы шагнем на 4 байта
    // т.е. если в матрице сделать шаг вверх или вниз на одну ячейку - то в памяти мы шагнем на так называемый stride=width*4 байта

    // TODO реализуйте этот кернел - просуммируйте две матрицы так чтобы получить максимально ПЛОХУЮ производительность с точки зрения memory coalesced паттерна доступа
}

namespace cuda {
void aplusb_matrix_bad(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32u &a, const gpu::gpu_mem_32u &b, gpu::gpu_mem_32u &c, unsigned int width, unsigned int height)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::aplusb_matrix_bad<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), b.cuptr(), c.cuptr(), width, height);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
